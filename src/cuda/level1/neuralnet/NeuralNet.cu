#include "cudacommon.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Timer.h"
#include "Utility.h"

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op)
{
    ;
}


// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//
// Arguments:
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op)
{
    ;
}
